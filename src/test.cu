#include <stdio.h>
//#include "../include/rnd_numbers.h"
#include "../include/test.h"
#include "../include/matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void test_rnd(bool verbose, size_t n) {
	size_t n2 = n*n;
	float *a, *b, *c;

	a = (float *) calloc(n2, sizeof(float));
	b = (float *) calloc(n2, sizeof(float));
	c = (float *) calloc(n2, sizeof(float));

	if (verbose) printf("Random ");
	random_matrix(n, a);
	if (verbose) printMat(n, a);

	if (verbose) printf("\nTransposed ");
	transpose(n, a, b);
	if (verbose) printMat(n, b);

	if (verbose) printf("\nFirst 2 Multiplied ");
	test_matmul(a,b,n,c);
	if (verbose) printMat(n, c);

	if (verbose) printf("\nFirst 2 Summed ");
	test_sum(a,b,n,c);
	if (verbose) printMat(n, c);
}
