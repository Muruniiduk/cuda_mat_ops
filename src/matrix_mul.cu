#include "hip/hip_runtime.h"
/* Basic Matrix Multiplication */
#include <stdio.h>
//#include "../include/rnd_numbers.h"
#include "../include/matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//See meetod on kopeeritud: https://github.com/slitvinov/cuda-examples/tree/master/day1/matmul
//Kasutab memory sharemist.
__global__ void matmul(const float* const a, const float* const b, const int n,
		float* const c) {
	// Base indexes inside A and B
	const int ia = (blockDim.y * blockIdx.y) * n;
	const int ib = blockDim.x * blockIdx.x;

	// Subindex inside a "tile"
	const int tileidx = n * threadIdx.y + threadIdx.x;

	// Index in C
	const int ic = ia + ib + tileidx;

	float sum = 0.0f;
	int aoff = 0, boff = 0;

	// Shared memory for the "tile" sub-matrix of A and B
	__shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

	// Go through "tiles" of size blockDim.x * blockDim.y
	for (; aoff < n; aoff += blockDim.x, boff += blockDim.y * n) {
		// Load the "tile" matrices from global memory to shared memory
		as[threadIdx.y][threadIdx.x] = a[ia + aoff + tileidx];
		bs[threadIdx.y][threadIdx.x] = b[ib + boff + tileidx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices
		for (int k = 0; k < BLOCK_SIZE; k++)
			sum += as[threadIdx.y][k] * bs[k][threadIdx.x];

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to global memory
	// each thread writes one element
	c[ic] = sum;
}

__global__ void sum(const float* const a, const float* const b, const int n,
		float* const c) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int n2 = n*n;
	// perform tid-th elements addition
	if (tid <  n2)
		c[tid] = a[tid] + b[tid];
}

void printMat(size_t n, float *data) {
	size_t i, j;
	printf("Matrix: \n");
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			printf("	%1.3f", data[i + n * j]);
		}
		printf("\n");
	}
}

void transpose(size_t n, float *data, float *newData) {
	size_t i, j;
//	size_t n2 = n * n;

	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			newData[j * n + i] = data[i * n + j];
		}
	}
}

void random_matrix(size_t n, float *hostData) {
	size_t n2 = n * n;
	hiprandGenerator_t gen;
	float *a;
	hipMalloc((void **) &a, n2 * sizeof(float));

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, 0);
	hiprandGenerateNormal(gen, a, n2, 0, 1);

	hipMemcpy(hostData, a, n2 * sizeof(float), hipMemcpyDeviceToHost);
//	printMat(n, hostData);
	hiprandDestroyGenerator(gen);
	hipFree(a);
}

void test_matmul(const float* const b, const float* const a, const int n,
		float* const c) {

	size_t n2 = n * n;
	float *da, *db, *dc;

	hipMalloc((void **) &da, n2 * sizeof(float));
	hipMalloc((void **) &db, n2 * sizeof(float));
	hipMalloc((void **) &dc, n2 * sizeof(float));
	hipMemcpy(da, (void **) a, n2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, (void **) b, n2 * sizeof(float), hipMemcpyHostToDevice);

//	hipMemcpy((void **) a, da, n2 * sizeof(float), hipMemcpyHostToDevice);
//	hipMemcpy((void **) b, db, n2 * sizeof(float), hipMemcpyHostToDevice);
//	int threadsPerBlock, blocksPerGrid;
//	if (n2 < 512) {
//		threadsPerBlock = n2;
//		blocksPerGrid = 1;
//	} else {
//		threadsPerBlock = 512;
//		blocksPerGrid = ceil(double(n2) / double(threadsPerBlock));
//	}
//	matmul<<<blocksPerGrid, threadsPerBlock>>>(da, db, n, dc);


	matmul<<<1028, 1>>>(da, db, n, dc);
	hipDeviceSynchronize();
	hipMemcpy(c, dc, n2 * sizeof(float), hipMemcpyDeviceToHost);
}

void test_sum(const float* const b, const float* const a, const int n,
		float* const c) {

	size_t n2 = n * n;
	float *da, *db, *dc;

	hipMalloc((void **) &da, n2 * sizeof(float));
	hipMalloc((void **) &db, n2 * sizeof(float));
	hipMalloc((void **) &dc, n2 * sizeof(float));
	hipMemcpy(da, (void **) a, n2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, (void **) b, n2 * sizeof(float), hipMemcpyHostToDevice);

	int threadsPerBlock, blocksPerGrid;

	// use 1 to 512 threads per block
	if (n2 < 512) {
		threadsPerBlock = n2;
		blocksPerGrid = 1;
	} else {
		threadsPerBlock = 512;
		blocksPerGrid = ceil(double(n2) / double(threadsPerBlock));
	}

//	sum<<<32, 1>>>(da, db, n, dc);
	sum<<<blocksPerGrid, threadsPerBlock>>>(da, db, n, dc);
	hipDeviceSynchronize();
	hipMemcpy(c, dc, n2 * sizeof(float), hipMemcpyDeviceToHost);
}

void my_mat_mul() {
	size_t n = 3;
	size_t n2 = 9;
	hiprandGenerator_t gen;
	float *a, *b, *c, *hostData;
	float *hostA, *hostB;

	hostData = (float *) calloc(n2, sizeof(float));

	hostA = (float *) calloc(n2, sizeof(float));
	hostB = (float *) calloc(n2, sizeof(float));

	hipMalloc((void **) &a, n2 * sizeof(float));
	hipMalloc((void **) &b, n2 * sizeof(float));
	hipMalloc((void **) &c, n2 * sizeof(float));
	printf("Allocated memory in GPU\n");

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

	hiprandSetPseudoRandomGeneratorSeed(gen, 0);

	hiprandGenerateNormal(gen, a, n2, 0, 1);
	hiprandGenerateNormal(gen, b, n2, 0, 1);
	printf("Random 3x3 matrices created\n");

	hipMemcpy(hostA, a, n2 * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hostB, b, n2 * sizeof(float), hipMemcpyDeviceToHost);

	printf("A ");
	printMat(n, hostA);
	printf("B ");
	printMat(n, hostB);

	matmul<<<32, 1>>>(a, b, n, c);
	hipDeviceSynchronize();

	hipMemcpy(hostData, c, n2 * sizeof(float), hipMemcpyDeviceToHost);

	printf("RESULT ");
	printMat(n, hostData);

	float* newData;
	newData = (float *) calloc(n2, sizeof(float));
	transpose(n, hostData, newData);
	printf("TRANSPOSE ");
	printMat(n, newData);

	hiprandDestroyGenerator(gen);
	hipFree(a);
	hipFree(b);
	hipFree(c);
	free(hostData);
	free(hostA);
	free(hostB);
}
